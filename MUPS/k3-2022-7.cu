#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void laplace(float* error, float** A, float** Anew, int n, int m) {
    __shared__ float sA[34][34];
    __shared__ float sAnew[32][32];
    __shared__ float serror[32][32];
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int localX = threadIdx.x + 1;
    int localY = threadIdx.y + 1;
    sA[threadIdx.x+1][threadIdx.y+1] = A[x][y];
    if (threadIdx.x == 0 && x != 0) {
        sA[localX-1][localY] = A[x-1][y];
    }
    if (threadIdx.x == blockDim.x-1 && x != m-1) {
        sA[localX+1][localY] = A[x+1][y];
    }
    if (threadIdx.y == 0 && y != 0) {
        sA[localX][localY-1] = A[x][y-1];
    }
    if (threadIdx.y == blockDim.y-1 && y != n-1) {
        sA[localX][localY+1] = A[x][y+1];
    }
    __syncthreads();
    sAnew[threadIdx.x][threadIdx.y] = (sA[localX-1][localY] + sA[localX+1][localY] + sA[localX][localY-1] + sA[localX][localY+1]) / 4;
    serror[threadIdx.x][threadIdx.y] = abs(sAnew[threadIdx.x][threadIdx.y] - sA[localX][localY]);
    __syncthreads();
    if (threadIdx.x == 0) {
        float maxError = 0.0;
        for (int i = 0; i < blockDim.x; ++i) {
            maxError = max(maxError, serror[i][threadIdx.y]);
        }
        serror[threadIdx.x][threadIdx.y] = maxError;
        if (threadIdx.y == 0) {
            maxError = 0.0;
            for (int i = 0; i < blockDim.y; ++i) {
                maxError = max(maxError, serror[threadIdx.x][i]);
            }
            atomicMax(error, maxError);
        }
    }
    Anew[x][y] = sAnew[threadIdx.x][threadIdx.y];
}

int main() {
    // ...
    while (error < tol && iter < iter_max) {
        // ...
        dim3 dimBlock(32, 32);
        dim3 dimGrid(ceil(m/32), ceil(n/32));
        laplace<<<dimBlock, dimGrid>>>(&gpuError, gpuA, gpuAnew, n, m);
        float** gpuAold = gpuA;
        gpuA = gpuAnew;
        gpuAnew = gpuAold;
        // ...
    }
    // ...
    return 0;
}
